#include "hip/hip_runtime.h"
#include "matrix.h"
#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hiprand/hiprand_kernel.h"

__global__ void cuda_kernel_rand_matrix(float *v, int length, int offset) {
    uint32_t id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= length) return;
    hiprandState curand_state;
    hiprand_init(0, id, offset, &curand_state);
    v[id] = hiprand_uniform(&curand_state);
}

void Matrix::Random() {
    int length = n_ * m_;
    float *dev_v;
    hipMalloc(&dev_v, sizeof(float) * n_ * m_);
    static int offset = 0;
    cuda_kernel_rand_matrix<<<(length + 127) / 128, 128>>>(dev_v, length, offset);
    hipMemcpy(v_, dev_v, sizeof(float) * n_ * m_, hipMemcpyDeviceToHost);
    hipFree(dev_v);
    offset++;
}

Matrix Matrix::operator*(const Matrix &mat) const {
    auto result = Matrix(n_, mat.m_);
    auto *buffer = new float[m_];
    for (int i = 0; i < n_; i++) {
        for (int k = 0; k < m_; k++) {
            buffer[k] = operator()(k, i);
        }
        for (int j = 0; j < mat.m_; j++) {
            float ans = 0.0f;
            auto *col = mat.GetBuffer() + j * m_;
            for (int k = 0; k < m_; k++) {
                ans += buffer[k] * col[k];
            }
            result(j, i) = ans;
        }
    }
    delete[] buffer;
    return result;
}

__global__ void cuda_kernel_matrix_multiplication(float *dev_res, float *dev_a, float *dev_b, int n, int m, int l) {
    uint32_t idy = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t idx = blockIdx.y * blockDim.y + threadIdx.y;
    uint32_t id = idx * n + idy;

    if (idy >= n || idx >= l) {
        return;
    }
    float ans = 0.0f;
    for (int k = 0; k < m; k++) {
        ans += dev_a[k * n + idy] * dev_b[idx * m + k];
    }
    dev_res[id] = ans;
}

Matrix MatrixMultiplication(const Matrix &m1, const Matrix &m2) {
    auto result = Matrix(m1.n_, m2.m_);
    float *dev_res;
    float *dev_a;
    float *dev_b;
    hipMalloc(&dev_res, sizeof(float) * m1.n_ * m2.m_);
    hipMalloc(&dev_a, sizeof(float) * m1.n_ * m1.m_);
    hipMalloc(&dev_b, sizeof(float) * m2.n_ * m2.m_);
    hipMemcpy(dev_a, m1.GetBuffer(), sizeof(float) * m1.n_ * m1.m_, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, m2.GetBuffer(), sizeof(float) * m2.n_ * m2.m_, hipMemcpyHostToDevice);
    cuda_kernel_matrix_multiplication<<<dim3((m1.n_ + 15) / 16, (m2.m_ + 15) / 16, 1), dim3(16, 16, 1)>>>(dev_res,
                                                                                                          dev_a, dev_b,
                                                                                                          m1.n_, m1.m_,
                                                                                                          m2.m_);
    hipMemcpy(result.GetBuffer(), dev_res, sizeof(float) * m1.n_ * m2.m_, hipMemcpyDeviceToHost);
    hipFree(dev_res);
    hipFree(dev_a);
    hipFree(dev_b);
    return result;
}

#define BUFFER_LENGTH 352

__global__ void
cuda_kernel_matrix_multiplication_shared(float *dev_res, float *dev_a, float *dev_b, int n, int m, int l) {
    uint32_t idy = blockIdx.x * blockDim.x + threadIdx.x;
    __shared__ float a_shared_buffer[BUFFER_LENGTH][17];
    __shared__ float b_shared_buffer[BUFFER_LENGTH][17];
    float ans;
    for (int k = 0; k < m; k += BUFFER_LENGTH) {
        for (int i = 0; i < BUFFER_LENGTH; i += 16) {
            if (k + i + threadIdx.y < m && idy < n) {
                a_shared_buffer[i + threadIdx.y][threadIdx.x] = dev_a[(k + i + threadIdx.y) * n + idy];
            } else {
                a_shared_buffer[i + threadIdx.y][threadIdx.x] = 0.0f;
            }
        }
        __syncthreads();
        for (int i = 0; i < l; i += 16) {
            int idx = i + threadIdx.y;
            for (int j = 0; j < BUFFER_LENGTH; j += 16) {
                if (k + j + threadIdx.x < m && idx < l) {
                    b_shared_buffer[j + threadIdx.x][threadIdx.y] = dev_b[idx * m + (k + j + threadIdx.x)];
                } else {
                    b_shared_buffer[j + threadIdx.x][threadIdx.y] = 0.0f;
                }
            }
            __syncthreads();
            if (idx < l && idy < n) {
                ans = 0.0f;
                for (int j = 0; j < BUFFER_LENGTH && k + j < m; j++) {
                    ans += a_shared_buffer[j][threadIdx.x] * b_shared_buffer[j][threadIdx.y];
                }
                __syncthreads();
                dev_res[idx * n + idy] += ans;
            }
        }
    }
}

Matrix MatrixMultiplicationShared(const Matrix &m1, const Matrix &m2) {
    auto result = Matrix(m1.n_, m2.m_);
    float *dev_res;
    float *dev_a;
    float *dev_b;
    hipMalloc(&dev_res, sizeof(float) * m1.n_ * m2.m_);
    hipMalloc(&dev_a, sizeof(float) * m1.n_ * m1.m_);
    hipMalloc(&dev_b, sizeof(float) * m2.n_ * m2.m_);
    hipMemcpy(dev_a, m1.GetBuffer(), sizeof(float) * m1.n_ * m1.m_, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, m2.GetBuffer(), sizeof(float) * m2.n_ * m2.m_, hipMemcpyHostToDevice);
    hipMemset(dev_res, 0, sizeof(float) * m1.n_ * m2.m_);
    cuda_kernel_matrix_multiplication_shared<<<dim3((m1.n_ + 15) / 16, 1, 1), dim3(16, 16, 1)>>>(dev_res, dev_a, dev_b,
                                                                                                 m1.n_, m1.m_, m2.m_);
    hipMemcpy(result.GetBuffer(), dev_res, sizeof(float) * m1.n_ * m2.m_, hipMemcpyDeviceToHost);
    hipFree(dev_res);
    hipFree(dev_a);
    hipFree(dev_b);
    return result;
}

float MaxDifference(const Matrix &m1, const Matrix &m2) {
    float res = 0.0f;
    for (int y = 0; y < m1.n_; y++) {
        for (int x = 0; x < m1.m_; x++) {
            res = std::max(res, std::abs(m1(x, y) - m2(x, y)));
        }
    }
    return res;
}

Matrix::Matrix() {
    n_ = 0;
    m_ = 0;
    v_ = nullptr;
}

Matrix MatrixMultiplicationCUBLAS(const hipblasHandle_t &handle, const Matrix &m1, const Matrix &m2) {
    auto result = Matrix(m1.n_, m2.m_);
    float *dev_res;
    float *dev_a;
    float *dev_b;
    hipMalloc(&dev_res, sizeof(float) * m1.n_ * m2.m_);
    hipMalloc(&dev_a, sizeof(float) * m1.n_ * m1.m_);
    hipMalloc(&dev_b, sizeof(float) * m2.n_ * m2.m_);
    hipMemcpy(dev_a, m1.GetBuffer(), sizeof(float) * m1.n_ * m1.m_, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, m2.GetBuffer(), sizeof(float) * m2.n_ * m2.m_, hipMemcpyHostToDevice);
    hipMemset(dev_res, 0, sizeof(float) * m1.n_ * m2.m_);
    float alpha = 1.0f;
    float beta = 0.0f;
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m1.n_, m2.m_, m1.m_, &alpha, dev_a, m1.n_, dev_b, m2.n_, &beta,
                dev_res, m1.n_);
    hipMemcpy(result.GetBuffer(), dev_res, sizeof(float) * m1.n_ * m2.m_, hipMemcpyDeviceToHost);
    hipFree(dev_res);
    hipFree(dev_a);
    hipFree(dev_b);
    return result;
}

__global__ void add_kernel(const int *A, const int *B, int *C) {
    C[threadIdx.x] = A[threadIdx.x] + B[threadIdx.x];
}

int main() {
    int A[] = {1, 2, 3, 4, 5};
    int B[] = {10, 20, 30, 40, 50};
    int C[5];
    int *dev_A, *dev_B, *dev_C;
    hipMalloc(&dev_A, sizeof(float) * 5);
    hipMalloc(&dev_B, sizeof(float) * 5);
    hipMalloc(&dev_C, sizeof(float) * 5);
    hipMemcpy(dev_A, A, sizeof(float) * 5, hipMemcpyHostToDevice);
    hipMemcpy(dev_B, B, sizeof(float) * 5, hipMemcpyHostToDevice);
    add_kernel<<<1, 5>>>(dev_A, dev_B, dev_C);
    hipMemcpy(C, dev_C, sizeof(float) * 5, hipMemcpyDeviceToHost);
    hipFree(dev_A);
    hipFree(dev_B);
    hipFree(dev_C);
    for (int i = 0; i < 5; i++) {
        printf("%d, ", C[i]);
    }
}